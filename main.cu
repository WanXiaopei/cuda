#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <time.h>
#include <iostream>
#include <algorithm>

#define M 4096
#define N 4096
#define K 4096

__host__ void init_matrix(float* m, size_t n) {
    for (size_t i = 0; i < n; i ++) {
        m[i] = float(rand() % 110) / 55 - 1.0f;
    }
}

__host__ int test_validation(float* rhc, float* thc, size_t n) {
    for (size_t i = 0; i < n; i ++) {
        if (std::abs(rhc[i] - thc[i]) > 0.0001) {
            std::cerr << "rhc[" << i << "]: " << rhc[i] << " vs. thc[" << i << "]: " << thc[i] << std::endl;
            return 1;
        }
    }
    return 0;
}

int capture_cuda_error() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << hipGetErrorName(error) << ": " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    return 0;
}

#define VALUE_CHECK(RHC, THC, N, NAME)                                  \
    if (test_validation(RHC, THC, N) != 0) {                            \
        std::cerr << #NAME << ": validation test failed." << std::endl; \
        return 1;                                                       \
    }                                                                   \
    std::cout << #NAME << ": validation test success." << std::endl;

#define CUDA_CHECK_STATUS()             \
    if (capture_cuda_error() != 0) {    \
        return 1;                       \
    }

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

#define TEST_SGEMM_MODULE(module, gdim, bdim)                                                       \
    {                                                                                               \
        std::cout << #module << ": " << std::endl;                                                  \
        float* tmpc;                                                                                \
        hipMalloc((void**)&tmpc, M * N * sizeof(float));                                           \
        {                                                                                           \
            TIME_GUARD(TEST_SGEMM_MODULE);                                                          \
            module<<<gdim, bdim>>>(M, N, K, alpha, a, b, beta, tmpc);                               \
            CUDA_CHECK_STATUS();                                                                    \
            hipDeviceSynchronize();                                                                \
        }                                                                                           \
        CUDA_CHECK_STATUS();                                                                        \
        hipMemcpy(thc, tmpc, M * N * sizeof(float), hipMemcpyDeviceToHost);                       \
        VALUE_CHECK(rhc, thc, (M * N), module);                                                     \
        hipFree(tmpc);                                                                             \
    }

int main() {
    srand(time(NULL));
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *a, *b, *c;
    float *ha = (float*)malloc(M * K * sizeof(float));
    float *hb = (float*)malloc(K * N * sizeof(float));
    float *rhc = (float*)malloc(M * N * sizeof(float));    // results of hipblasSgemm
    float *thc = (float*)malloc(M * N * sizeof(float));    // results of my kernels
    
    hipMalloc((void**)&a, M * K * sizeof(float));
    hipMalloc((void**)&b, K * N * sizeof(float));
    hipMalloc((void**)&c, M * N * sizeof(float));

    {
        TIME_GUARD(INIT);
        init_matrix(ha, M * K);
        init_matrix(hb, K * N);
        hipMemcpy(a, ha, M * K * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(b, hb, K * N * sizeof(float), hipMemcpyHostToDevice);
        CUDA_CHECK_STATUS();
    }

    hipDeviceSynchronize();
    // warmup the machine
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b, N, a, K, &beta, c, N);
    hipDeviceSynchronize();
    {
        TIME_GUARD(CUBLAS);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, b, N, a, K, &beta, c, N);
        hipDeviceSynchronize();
        CUDA_CHECK_STATUS();
    }
    hipMemcpy(rhc, c, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(a);
    hipFree(b);
    hipFree(c);
    free(ha);
    free(hb);
    free(rhc);
    free(thc);

    return 0;
}
